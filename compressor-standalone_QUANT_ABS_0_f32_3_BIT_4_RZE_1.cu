#include "hip/hip_runtime.h"
/*
This file is part of the LC framework for synthesizing high-speed parallel lossless and error-bounded lossy data compression and decompression algorithms for CPUs and GPUs.

BSD 3-Clause License

Copyright (c) 2021-2024, Noushin Azami, Alex Fallin, Brandon Burtchell, Andrew Rodriguez, Benila Jerald, Yiqian Liu, and Martin Burtscher
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

URL: The latest version of this code is available at https://github.com/burtscher/LC-framework.

Sponsor: This code is based upon work supported by the U.S. Department of Energy, Office of Science, Office of Advanced Scientific Research (ASCR), under contract DE-SC0022223.
*/


#define NDEBUG

using byte = unsigned char;
static const int CS = 1024 * 16;  // chunk size (in bytes) [must be multiple of 8]
//static const int CS = 384*352 / 8;  // fraction of detector size 
//static const int CS = 384*352 / 16;

//static const int TPB = 768;  // threads per block [must be power of 2 and at least 128]
//static const int TPB = 512;  // threads per block [must be power of 2 and at least 128]
//static const int TPB = 768;  // threads per block [must be power of 2 and at least 128]

//static const int TPB = 768-32;  // threads per block [must be power of 2 and at least 128]

static const int TPB = 512-32;  // threads per block [must be power of 2 and at least 128]

#if defined(__AMDGCN_WAVEFRONT_SIZE) && (__AMDGCN_WAVEFRONT_SIZE == 64)
#define WS 64
#else
#define WS 32
#endif

#define NUMSTREAMS 4
#define NUMSEGS 1

#include <string>
#include <cmath>
#include <cassert>
#include <stdexcept>
#include <hip/hip_runtime.h>
#include "include/sum_reduction.h"
#include "include/max_scan.h"
#include "include/prefix_sum.h"
#include "preprocessors/d_QUANT_ABS_0_f32.h"
#include "components/d_BIT_4.h"
#include "components/d_RZE_1.h"


// copy (len) bytes from shared memory (source) to global memory (destination)
// source must we word aligned
static inline __device__ void s2g(void* const __restrict__ destination, const void* const __restrict__ source, const int len)
{
  const int tid = threadIdx.x;
  const byte* const __restrict__ input = (byte*)source;
  byte* const __restrict__ output = (byte*)destination;
  if (len < 128) {
    if (tid < len) output[tid] = input[tid];
  } else {
    const int nonaligned = (int)(size_t)output;
    const int wordaligned = (nonaligned + 3) & ~3;
    const int linealigned = (nonaligned + 127) & ~127;
    const int bcnt = wordaligned - nonaligned;
    const int wcnt = (linealigned - wordaligned) / 4;
    const int* const __restrict__ in_w = (int*)input;
    if (bcnt == 0) {
      int* const __restrict__ out_w = (int*)output;
      if (tid < wcnt) out_w[tid] = in_w[tid];
      for (int i = tid + wcnt; i < len / 4; i += TPB) {
        out_w[i] = in_w[i];
      }
      if (tid < (len & 3)) {
        const int i = len - 1 - tid;
        output[i] = input[i];
      }
    } else {
      const int shift = bcnt * 8;
      const int rlen = len - bcnt;
      int* const __restrict__ out_w = (int*)&output[bcnt];
      if (tid < bcnt) output[tid] = input[tid];
      if (tid < wcnt) out_w[tid] = __funnelshift_r(in_w[tid], in_w[tid + 1], shift);
      for (int i = tid + wcnt; i < rlen / 4; i += TPB) {
        out_w[i] = __funnelshift_r(in_w[i], in_w[i + 1], shift);
      }
      if (tid < (rlen & 3)) {
        const int i = len - 1 - tid;
        output[i] = input[i];
      }
    }
  }
}


static __device__ int g_chunk_counter[NUMSTREAMS];


static __global__ void d_reset(const short stream_num)
{
  g_chunk_counter[stream_num] = 0;
}


static inline __device__ void propagate_carry(const int value, const int chunkID, volatile int* const __restrict__ fullcarry, int* const __restrict__ s_fullc)
{
  if (threadIdx.x == TPB - 1) {  // last thread
    fullcarry[chunkID] = (chunkID == 0) ? value : -value;
  }

  if (chunkID != 0) {
    if (threadIdx.x + WS >= TPB) {  // last warp
      const int lane = threadIdx.x % WS;
      const int cidm1ml = chunkID - 1 - lane;
      int val = -1;
      __syncwarp();  // not optional
      do {
        if (cidm1ml >= 0) {
          val = fullcarry[cidm1ml];
        }
      } while ((__any_sync(~0, val == 0)) || (__all_sync(~0, val <= 0)));
#if defined(WS) && (WS == 64)
      const long long mask = __ballot_sync(~0, val > 0);
      const int pos = __ffsll(mask) - 1;
#else
      const int mask = __ballot_sync(~0, val > 0);
      const int pos = __ffs(mask) - 1;
#endif
      int partc = (lane < pos) ? -val : 0;
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 800)
      partc = __reduce_add_sync(~0, partc);
#else
      partc += __shfl_xor_sync(~0, partc, 1);
      partc += __shfl_xor_sync(~0, partc, 2);
      partc += __shfl_xor_sync(~0, partc, 4);
      partc += __shfl_xor_sync(~0, partc, 8);
      partc += __shfl_xor_sync(~0, partc, 16);
#endif
      if (lane == pos) {
        const int fullc = partc + val;
        fullcarry[chunkID] = fullc + value;
        *s_fullc = fullc;
      }
    }
  }
}


#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ == 800)
static __global__ __launch_bounds__(TPB, 3)
#else
static __global__ __launch_bounds__(TPB, 2)
#endif
  void d_encode(const byte* const __restrict__ input, const int insize, byte* const __restrict__ output, int* const __restrict__ outsize, int* const __restrict__ fullcarry, const short stream_num)
{
  // allocate shared memory buffer
  __shared__ long long chunk [3 * (CS / sizeof(long long))];

  // split into 3 shared memory buffers
  byte* in = (byte*)&chunk[0 * (CS / sizeof(long long))];
  byte* out = (byte*)&chunk[1 * (CS / sizeof(long long))];
  byte* const temp = (byte*)&chunk[2 * (CS / sizeof(long long))];

  // initialize
  const int tid = threadIdx.x;
  const int last = 3 * (CS / sizeof(long long)) - 2 - WS;
  const int chunks = (insize + CS - 1) / CS;  // round up
  int* const head_out = (int*)output;
  unsigned short* const size_out = (unsigned short*)&head_out[1];
  byte* const data_out = (byte*)&size_out[chunks];

  // loop over chunks
  do {
    // assign work dynamically
    if (tid == 0) chunk[last] = atomicAdd(&g_chunk_counter[stream_num], 1);
    __syncthreads();  // chunk[last] produced, chunk consumed

    // terminate if done
    const int chunkID = chunk[last];
    const int base = chunkID * CS;
    if (base >= insize) break;

    // load chunk
    const int osize = min(CS, insize - base);
    long long* const input_l = (long long*)&input[base];
    long long* const out_l = (long long*)out;
    for (int i = tid; i < osize / 8; i += TPB) {
      out_l[i] = input_l[i];
    }
    const int extra = osize % 8;
    if (tid < extra) out[osize - extra + tid] = input[base + osize - extra + tid];

    // encode chunk
    __syncthreads();  // chunk produced, chunk[last] consumed
    int csize = osize;
    bool good = true;
    if (good) {
      byte* tmp = in; in = out; out = tmp;
      good = d_BIT_4(csize, in, out, temp);
     __syncthreads();
    }
    if (good) {
      byte* tmp = in; in = out; out = tmp;
      good = d_RZE_1(csize, in, out, temp);
     __syncthreads();
    }

    // handle carry
    if (!good || (csize >= osize)) csize = osize;
    propagate_carry(csize, chunkID, fullcarry, (int*)temp);

    // reload chunk if incompressible
    if (tid == 0) size_out[chunkID] = csize;
    if (csize == osize) {
      // store original data
      long long* const out_l = (long long*)out;
      for (int i = tid; i < osize / 8; i += TPB) {
        out_l[i] = input_l[i];
      }
      const int extra = osize % 8;
      if (tid < extra) out[osize - extra + tid] = input[base + osize - extra + tid];
    }
    __syncthreads();  // "out" done, temp produced

    // store chunk
    const int offs = (chunkID == 0) ? 0 : *((int*)temp);
    s2g(&data_out[offs], out, csize);

    // finalize if last chunk
    if ((tid == 0) && (base + CS >= insize)) {
      // output header
      head_out[0] = insize;
      // compute compressed size
      *outsize = &data_out[fullcarry[chunkID]] - output;
    }
  } while (true);
}


struct GPUTimer
{
  hipEvent_t beg, end;
  GPUTimer() {hipEventCreate(&beg); hipEventCreate(&end);}
  ~GPUTimer() {hipEventDestroy(beg); hipEventDestroy(end);}
  void start() {hipEventRecord(beg, 0);}
  double stop() {hipEventRecord(end, 0); hipEventSynchronize(end); float ms; hipEventElapsedTime(&ms, beg, end); return 0.001 * ms;}
};


static void CheckCuda(const int line)
{
  hipError_t e;
  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "CUDA error %d on line %d: %s\n\n", e, line, hipGetErrorString(e));
    throw std::runtime_error("LC error");
  }
}


int main(int argc, char* argv [])
{
  printf("GPU LC 1.2 Algorithm: QUANT_ABS_0_f32 BIT_4 RZE_1\n");
  printf("Copyright 2024 Texas State University\n\n");
  // read input from file
  //if (argc < 3) {printf("USAGE: %s input_file_name compressed_file_name [performance_analysis (y)]\n\n", argv[0]); return -1;}
  if (argc < 4) {
    printf("USAGE: %s input_file_name compressed_file_name num_events [performance_analysis (y)]\n\n",
           argv[0]); 
    return -1;
  }
  char* leftover;
  int num_events = strtol(argv[3], &leftover, 10);
  FILE* const fin = fopen(argv[1], "rb");
  // read input from file
  fseek(fin, 0, SEEK_END);
  const int fsize = ftell(fin);  assert(fsize > 0);
  byte* const input = new byte [fsize];
  fseek(fin, 0, SEEK_SET);
  const int original_insize = fread(input, 1, fsize, fin);  assert(original_insize == fsize);
  fclose(fin);
  printf("original size: %d bytes\n", original_insize);
  printf("original size: %d bytes\n", original_insize);
  printf("num events: %d \n", num_events);

  const int insize = original_insize / num_events;
  printf("Bytes per event: %d\n", insize);

  // Check if the third argument is "y" to enable performance analysis
  char* perf_str = argv[4];
  //char* perf_str = argv[3];
  bool perf = false;
  if (perf_str != nullptr && strcmp(perf_str, "y") == 0) {
    perf = true;
  } else if (perf_str != nullptr && strcmp(perf_str, "y") != 0) {
    fprintf(stderr, "ERROR: Invalid argument. Use 'y' or nothing.\n");
    throw std::runtime_error("LC error");
  }

  // get GPU info
  hipSetDevice(0);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  if ((deviceProp.major == 9999) && (deviceProp.minor == 9999)) {fprintf(stderr, "ERROR: no CUDA capable device detected\n\n"); throw std::runtime_error("LC error");}
  //const int SMs = deviceProp.multiProcessorCount;
  //const int mTpSM = deviceProp.maxThreadsPerMultiProcessor;
  const int blocks = 24;//SMs * (mTpSM / TPB);
  //const int blocks = 24;//SMs * (mTpSM / TPB);

  const int chunks = (insize + CS - 1) / CS;  // round up
  CheckCuda(__LINE__);
  const int maxsize = 3 * sizeof(int) + chunks * sizeof(short) + chunks * CS;

  hipStream_t streams[NUMSTREAMS];

  // allocate GPU memory
  byte* dencoded[NUMSTREAMS];
  byte* d_input[NUMSTREAMS];
  byte* d_encoded[NUMSTREAMS];
  int* d_encsize[NUMSTREAMS];
  byte* dpreencdata[NUMSTREAMS];
  //int dpreencsize = insize; // Assume same input size for every event/stream
  int dpreencsize = insize/NUMSEGS; // Divide frames into segments 
  int evts_per_stream = num_events / NUMSTREAMS * NUMSEGS;
  for (size_t i=0; i < NUMSTREAMS; ++i) {
    hipHostMalloc((void **)&dencoded[i], maxsize);
    //hipMalloc((void **)&d_input[i], insize*evts_per_stream);
    //hipMemcpy(d_input[i], input + i*insize*evts_per_stream, insize*evts_per_stream, hipMemcpyHostToDevice);
    hipMalloc((void **)&d_input[i], dpreencsize*evts_per_stream);
    //hipMemcpy(d_input[i], input + i*dpreencsize*evts_per_stream, insize*evts_per_stream, hipMemcpyHostToDevice);
    hipMemcpy(d_input[i], input + i*dpreencsize*evts_per_stream, dpreencsize*evts_per_stream, hipMemcpyHostToDevice);
    hipMalloc((void **)&d_encoded[i], maxsize);
    hipMalloc((void **)&d_encsize[i], sizeof(int));
    CheckCuda(__LINE__);
    //hipMalloc((void **)&dpreencdata[i], insize*evts_per_stream);
    //hipMemcpy(dpreencdata[i], d_input[i], insize*evts_per_stream, hipMemcpyDeviceToDevice);
    hipMalloc((void **)&dpreencdata[i], dpreencsize*evts_per_stream);
    hipMemcpy(dpreencdata[i], d_input[i], dpreencsize*evts_per_stream, hipMemcpyDeviceToDevice);
    hipStreamCreate(&streams[i]);
  }

  if (perf) {
    //int* d_fullcarry[NUMSTREAMS];
    for (size_t i=0; i < NUMSTREAMS; ++i) {
      byte* d_preencdata;
      hipMalloc((void **)&d_preencdata, insize);
      hipMemcpy(d_preencdata, d_input[i], insize, hipMemcpyDeviceToDevice);
      int dpreencsize = insize;
      double paramv[] = {3}; 
      d_QUANT_ABS_0_f32(dpreencsize, d_preencdata, 1, paramv);
//      d_QUANT_ABS_0_f32(dpreencsize, d_preencdata, 1, paramv);
//      byte* d_ptr = dpreencdata[i] + dpreencsize*evt;
//      d_QUANT_ABS_0_f32_stream(dpreencsize, d_ptr, 1, paramv, streams[i]);

      hipFree(d_preencdata);
    }
  }


  int* d_fullcarry[NUMSTREAMS];


  for (size_t i=0; i < NUMSTREAMS; ++i)
    hipMalloc((void **)&d_fullcarry[i], chunks * sizeof(int));

  hipDeviceSynchronize();
  GPUTimer dtimer;
  double paramv[] = {3};
  dtimer.start();

  for (size_t evt=0; evt < evts_per_stream; ++evt) {
    for (size_t i=0; i < NUMSTREAMS; ++i) {
      byte* d_ptr = dpreencdata[i] + dpreencsize*evt;
      //d_QUANT_ABS_0_f32(dpreencsize, d_ptr, 1, paramv);
      d_QUANT_ABS_0_f32_stream(dpreencsize, d_ptr, 1, paramv, streams[i]);

      d_reset<<<1, 1, 0, streams[i]>>>(i);
      //hipMemset(d_fullcarry[i], 0, chunks * sizeof(byte));
      hipMemsetAsync(d_fullcarry[i], 0, chunks * sizeof(byte), streams[i]);
      d_encode<<<blocks, TPB, 0, streams[i]>>>(dpreencdata[i]+dpreencsize*evt, dpreencsize, d_encoded[i], d_encsize[i], d_fullcarry[i], i);
    }
  }
  hipDeviceSynchronize();
    //hipMalloc((void **)&d_fullcarry[i], chunks * sizeof(int));

  double runtime = dtimer.stop();

  for (size_t i=0; i < NUMSTREAMS; ++i)
    hipFree(d_fullcarry[i]);

  // get encoded GPU result
  //int dencsize = 0;
  //hipMemcpy(&dencsize, d_encsize, sizeof(int), hipMemcpyDeviceToHost);
  //hipMemcpy(dencoded, d_encoded, dencsize, hipMemcpyDeviceToHost);
  //printf("encoded size: %d bytes\n", dencsize);
  //CheckCuda(__LINE__);

  //const float CR = (100.0 * dencsize) / insize;
  //printf("ratio: %6.2f%% %7.3fx\n", CR, 100.0 / CR);

  // calculate theoretical occupancy
  int maxActiveBlocks;
  hipOccupancyMaxActiveBlocksPerMultiprocessor( &maxActiveBlocks, 
                                                 d_encode, TPB, 
                                                 0);

  int device;
  hipDeviceProp_t props;
  hipGetDevice(&device);
  hipGetDeviceProperties(&props, device);

  float occupancy = (maxActiveBlocks * TPB / props.warpSize) / 
                    (float)(props.maxThreadsPerMultiProcessor / 
                            props.warpSize);

  printf("Launched blocks of size %d. Theoretical occupancy: %f\nMax active blocks:%d\n", 
         TPB, occupancy, maxActiveBlocks);

  if (perf) {
    printf("encoding time: %.6f s\n", runtime);
    double throughput = insize * 0.000000001 / runtime;
    printf("original size: %d \n", original_insize );

    printf("encoding throughput: %8.3f Gbytes/s\n", original_insize * 0.000000001 / runtime);
    printf("encoding throughput: %8.3f Gbytes/s\n", throughput*NUMSTREAMS*evts_per_stream/NUMSEGS);
    printf("\tNumber of streams: %d\n", NUMSTREAMS);
    printf("\tEvents per stream: %d\n", evts_per_stream);
    printf("\tNumber of Segments: %d\n", NUMSEGS);


    CheckCuda(__LINE__);
  }

  // write to file
  //FILE* const fout = fopen(argv[2], "wb");
  //fwrite(dencoded, 1, dencsize, fout);
  //fclose(fout);

  // clean up GPU memory
  for (size_t i=0; i < NUMSTREAMS; ++i) {
    hipFree(d_input[i]);
    hipFree(d_encoded[i]);
    hipFree(d_encsize[i]);
    CheckCuda(__LINE__);
  }

  // clean up
  delete [] input;
  hipHostFree(dencoded);
  return 0;
}
